#include <stdio.h>
#include <hip/hip_runtime.h>


// Imprime las propiedades del dispositivo.
void printDevProp(hipDeviceProp_t devProp) {
    printf("  Nombre del GPU:                %s\n",  devProp.name);
    printf("  Memoria global:                %.0f MBytes (%llu bytes)\n",
      (float)devProp.totalGlobalMem/1048576.0f, (unsigned long long) devProp.totalGlobalMem);
    printf("  Numero de multiprocesadores:   %d\n",  devProp.multiProcessorCount);
    return;
}

// Programa principal.
int main() {
    // Numero de dispositivos CUDA.
    int devCount;
    hipGetDeviceCount(&devCount);
    printf("Test NVIDIA CUDA...\n");
    printf("Existe %d GPU NVIDIA CUDA.\n", devCount);

    // Mostrar propiedades de todos los dispositivos.
    for (int i = 0; i < devCount; ++i) {
        // Obtener propiedades del dispositivos.
        printf("\nDispositivo CUDA #%d\n", i+1);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        printDevProp(devProp);
    }

    return 0;
}
